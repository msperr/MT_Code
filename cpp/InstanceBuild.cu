#include "hip/hip_runtime.h"
#include "Instance.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "simple_vector.h"
#include "simple_matrix.h"

__global__ void calcParetoRefuelpoints(
	int ds,
	int dt,
	int num_vertices,
	int num_refuelpoints,
	double cost_per_meter,
	double fuel_per_meter,
	double refuel_per_second,
	simple_vector_base<double> vertex_starttime,
	simple_vector_base<double> vertex_finishtime,
	simple_matrix_base<double> arc_time,
	simple_matrix_base<double> arc_dist,
	simple_matrix_base<bool> output
	)
{

	extern __shared__ unsigned char mem[];

	volatile bool* dominance = (bool*)mem;
	double* time = (double*)(dominance + 4 * blockDim.x * blockDim.y); // use more memory in order to avoid bank conflicts
	double* phi_0 = time + num_refuelpoints;
	double* phi_1 = phi_0 + num_refuelpoints;
	double* phi_2 = phi_1 + num_refuelpoints;
	double* phi_3 = phi_2 + num_refuelpoints;

	const int s = ds + blockIdx.x;
	const int t = dt + blockIdx.y;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	if (s < num_vertices && t < num_vertices) {

		if (vertex_finishtime(s) + arc_time(s, t) <= vertex_starttime(t)) {

			for (int p = ty * blockDim.x + tx; p < num_refuelpoints; p += blockDim.x * blockDim.y) {
				const int r = num_vertices + p;
				const double time_r = vertex_starttime(t) - arc_time(r, t) - arc_time(s, r) - vertex_finishtime(s);
				const double dist_s_r = arc_dist(s, r);
				const double dist_r_t = arc_dist(r, t);
				time[p] = time_r;
				phi_0[p] = cost_per_meter * (dist_s_r + dist_r_t);
				phi_1[p] = fuel_per_meter * dist_s_r + max(-min(refuel_per_second * time_r, 1.0) + fuel_per_meter * dist_r_t, 0.0);
				phi_2[p] = max(fuel_per_meter * dist_s_r - min(refuel_per_second * time_r, 1.0), 0.0) + fuel_per_meter * dist_r_t;
				phi_3[p] = fuel_per_meter * dist_s_r - min(refuel_per_second * time_r, 1.0) + fuel_per_meter * dist_r_t;
			}

			__syncthreads();

			for (int p = ty; p < num_refuelpoints; p += blockDim.y) {
				
				if (time[p] < 0.0) {
					if (!tx)
						output(blockIdx.y * num_vertices + s, p) = true;
				} else {

					bool dominated = false;
					for (int q = tx; q < num_refuelpoints; q += blockDim.x) {
						const bool dominance_p_q = phi_0[p] <= phi_0[q] && phi_1[p] <= phi_1[q] && phi_2[p] <= phi_2[q] && phi_3[p] <= phi_3[q];
						const bool dominance_q_p = phi_0[p] >= phi_0[q] && phi_1[p] >= phi_1[q] && phi_2[p] >= phi_2[q] && phi_3[p] >= phi_3[q];
						dominated |= dominance_q_p && (p < q || !dominance_p_q);
					}
					dominance[(ty * blockDim.x + tx) << 2] = dominated;

					// use warp sync
					if (blockDim.x == 32) {
						if (tx < 16) {
							dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + 16) << 2];
							dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + 8) << 2];
							dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + 4) << 2];
							dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + 2) << 2];
							dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + 1) << 2];
						}
					} else {
						if (tx < blockDim.x >> 1)
							for (unsigned int stride = blockDim.x >> 1; stride; stride >>= 1)
								dominance[(ty * blockDim.x + tx) << 2] |= dominance[(ty * blockDim.x + tx + stride) << 2];
					}

					if (!tx)
						output(blockIdx.y * num_vertices + s, p) = dominance[(ty * blockDim.x + tx) << 2];
				}
			}

		} else {
			for (int p = ty * blockDim.x + tx; p < num_refuelpoints; p += blockDim.x * blockDim.y)
				output(blockIdx.y * num_vertices + s, p) = true;
		}
	}
}

void Instance::build() {

	int devID = 0;

	hipError_t cuda_error;
	hipDeviceProp_t deviceProp;
	cuda_error = hipGetDevice(&devID);
	assert(cuda_error == hipSuccess);

	cuda_error = hipGetDeviceProperties(&deviceProp, devID);
	assert(cuda_error == hipSuccess);

	printf("Building instance on GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

	int block_size = (deviceProp.major < 2) ? 16 : 32;
	int grid_size = 32;

	const int num_vertices = num_vehicles + num_trips;

	simple_matrix<double, hipMemoryTypeDevice> d_arc_dist(arc_dist);
	simple_matrix<double, hipMemoryTypeDevice> d_arc_time(arc_time);
//	simple_vector<double, hipMemoryTypeDevice> d_vertex_starttime(vertex_starttime);
	simple_vector<double> d_vertex_starttime(vertex_starttime);
//	simple_vector<double, hipMemoryTypeDevice> d_vertex_finishtime(vertex_finishtime);
	simple_vector<double> d_vertex_finishtime(vertex_finishtime);

	assert(num_vertices * grid_size * num_refuelpoints < deviceProp.totalGlobalMem);
	simple_matrix<bool, hipMemoryTypeDevice> d_output(grid_size * num_vertices, num_refuelpoints);

	dim3 threads(block_size, block_size);
	dim3 grid(grid_size, grid_size);

	const int size = 4 * threads.x * threads.y * sizeof(bool) + 5 * num_refuelpoints * sizeof(double);
	assert(size < deviceProp.sharedMemPerBlock);



	int numel = 0;
	for (int t = 0; t < num_vertices; t++)
		for (int s = 0; s < num_vertices; s++)
			if ((vertex_customer(s) != vertex_customer(t)) && (vertex_finishtime(s) + arc_time(s, t) <= vertex_starttime(t)))
				numel++;

	arc_refuelpoints.resize(num_vertices, num_vertices, numel);


	StopWatchWin stopwatch;
	stopwatch.start();

	for (int dt = 0; dt < num_vertices; dt += grid_size) {
		for (int ds = 0; ds < num_vertices; ds += grid_size) {

			calcParetoRefuelpoints <<< grid, threads, size >>> (ds, dt, num_vertices, num_refuelpoints, cost_per_meter, fuel_per_meter, refuel_per_second, d_vertex_starttime, d_vertex_finishtime, d_arc_time, d_arc_dist, d_output);

			checkCudaErrors(hipPeekAtLastError());
			checkCudaErrors(hipDeviceSynchronize());

			const double percentage = (double) (dt * num_vertices + std::min(ds + grid_size, num_vertices) * std::min(grid_size, num_vertices - dt)) / num_vertices / num_vertices;
			printf("[%.*s%.*s] % 3.1f%% in %.1fs %d %d\r", 
				(int)(100 * percentage), "####################################################################################################",
				100 - (int)(100 * percentage), "                                                                                                    ",
				100.0 * percentage,
				stopwatch.getTime() / 1000.0, ds, dt);
		}

		simple_matrix<bool, hipMemoryTypeHost> output(d_output);

		for (int t = dt; t < dt + grid_size && t < num_vertices; t++) {
			arc_refuelpoints.appendRow();
			for (int s = 0; s < num_vertices; s++) {
				if ((vertex_customer(s) != vertex_customer(t)) && (vertex_finishtime(s) + arc_time(s, t) <= vertex_starttime(t))) {
					auto& refuelpoints = arc_refuelpoints.appendElement(s);
					for (int r = 0; r < num_refuelpoints; r++)
						if (!output((t - dt) * num_vehicles + s, r))
							refuelpoints.push_back(r);
				}
			}
		}
	}
	stopwatch.stop();
	printf("\n");
}